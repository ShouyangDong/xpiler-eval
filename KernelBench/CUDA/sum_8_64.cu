
#include <hip/hip_runtime.h>
// Kernel: reduce along axis=1 for input [8, 64] -> output [8]
// Each thread handles one row
__global__ void sum_kernel_dev(const float* __restrict__ input, float* __restrict__ output) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= 8) return;  // Only 8 rows

    float sum = 0.0f;
    for (int col = 0; col < 64; col++) {
        int idx = row * 64 + col;  // input[row][col]
        sum += input[idx];
    }
    output[row] = sum;  // No division for sum
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C" void sum_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 8 * 64;   // 512
        const int output_size = 8;       // 8

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(8);
        dim3 numBlocks(1);  // 8 threads → one block is enough

        sum_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);
        // Copy result back to host
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_input);
        hipFree(d_output);
}
