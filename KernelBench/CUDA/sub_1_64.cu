
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(64)
    sub(float *__restrict__ A, float *__restrict__ B, float *__restrict__ C) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 64) {
    C[idx] = A[idx] - B[idx];
  }
}

extern "C" void sub_kernel(float *h_A, float *h_B, float *h_C, int total) {
  float *d_A, *d_B, *d_C;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_B, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(64);
  dim3 numBlocks(1);

  sub<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
