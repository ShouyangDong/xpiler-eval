
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(64)
sub(float *__restrict__ A, float *__restrict__ B, float *__restrict__ C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 64) {
        C[idx] = A[idx] - B[idx];
    }
}

extern "C" void sub_kernel(float *h_A, float *h_B, float *h_C, int total) {
    float *d_A, *d_B, *d_C;

    // 分配设备内存
    hipMalloc(&d_A, total * sizeof(float));
    hipMalloc(&d_B, total * sizeof(float));
    hipMalloc(&d_C, total * sizeof(float));

    // Host → Device 数据拷贝
    hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total * sizeof(float), hipMemcpyHostToDevice);

    // 配置 kernel 启动参数
    dim3 blockSize(64); // 每个block有64个线程
    dim3 numBlocks(1);  // 只需要1个block来处理这64个元素

    // 启动 kernel
    sub<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    // Device → Host 数据拷贝
    hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
