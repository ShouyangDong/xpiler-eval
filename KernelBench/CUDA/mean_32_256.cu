
#include <hip/hip_runtime.h>


__global__ void mean_kernel_dev(const float *__restrict__ input,
                                float *__restrict__ output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col >= 256)
    return;

  float sum = 0.0f;
  for (int row = 0; row < 32; row++) {
    int idx = row * 256 + col;
    sum += input[idx];
  }
  output[col] = sum / 32.0f;
}

extern "C" void mean_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 32 * 256;
  const int output_size = 256;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(256);
  dim3 numBlocks(1);

  mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}