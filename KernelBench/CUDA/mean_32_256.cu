// Generated: mean along last dimension for input [4x3x5] -> [4x3]
// Total input: 60, Reduce size: 5, Output count: 12

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
mean(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 12) return;

    float sum = 0.0f;
    for (int i = 0; i < 5; i++) {
        int in_idx = out_idx * 5 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum / 5;  // mean = sum / N
}

extern "C" void mean_kernel_4_3_5(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 60;
    const int output_size = 12;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    mean<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
