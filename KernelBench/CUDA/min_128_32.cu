// Generated: min along last dimension for input [128x32] -> [128]
// Total input: 4096, Reduce size: 32, Output count: 128

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
min_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 128) return;

    float min_val = FLT_MAX;
    for (int i = 0; i < 32; i++) {
        int in_idx = out_idx * 32 + i;
        float val = input[in_idx];
        min_val = fminf(min_val, val);
    }
    output[out_idx] = min_val;
}

extern "C" void min(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 4096;
    const int output_size = 128;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    min_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
