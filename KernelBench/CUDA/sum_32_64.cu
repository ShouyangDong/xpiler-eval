// Generated: sum along last dimension for input [32x64] -> [32]
// Total input: 2048, Reduce size: 64, Output count: 32

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
sum_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 32) return;

    float sum = 0.0f;
    for (int i = 0; i < 64; i++) {
        int in_idx = out_idx * 64 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum;
}

extern "C" void sum(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 2048;
    const int output_size = 32;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    sum_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
