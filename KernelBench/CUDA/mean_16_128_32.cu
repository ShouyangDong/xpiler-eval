
#include <hip/hip_runtime.h>


__global__ void mean_kernel_dev(const float *__restrict__ input,
                                float *__restrict__ output) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int output_size = 16 * 32;
  if (idx >= output_size)
    return;

  int n = idx / 32;
  int w = idx % 32;

  float sum = 0.0f;
  for (int h = 0; h < 128; h++) {

    int in_idx = n * (128 * 32) + h * 32 + w;
    sum += input[in_idx];
  }
  output[idx] = sum / 128.0f;
}

extern "C" void mean_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 16 * 128 * 32;
  const int output_size = 16 * 32;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(512);
  dim3 numBlocks(1);

  mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}