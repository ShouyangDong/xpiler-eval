
#include <hip/hip_runtime.h>
// Kernel: reduce along axis=1 for input [16, 128, 32] -> output [16, 32]
// Each thread handles one (n, w) position
__global__ void mean_kernel_dev(const float* __restrict__ input, float* __restrict__ output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // linear index in [0, 511]
    const int output_size = 16 * 32;  // 512
    if (idx >= output_size) return;

    int n = idx / 32;  // batch index: 0~15
    int w = idx % 32;  // width index: 0~31

    float sum = 0.0f;
    for (int h = 0; h < 128; h++) {
        // input[n][h][w]
        int in_idx = n * (128 * 32) + h * 32 + w;
        sum += input[in_idx];
    }
    output[idx] = sum / 128.0f;  // Divide by reduction size (axis=1 has 128 elements)
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C" void mean_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 16 * 128 * 32;   // 65536
        const int output_size = 16 * 32;        // 512

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(512);
        dim3 numBlocks(1);  // 512 threads → one block is enough

        mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

        // Copy result back to host
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_input);
        hipFree(d_output);
}