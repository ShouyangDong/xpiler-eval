
#include <hip/hip_runtime.h>


constexpr int N = 8;
constexpr int C = 768;
constexpr int H = 1;
constexpr int W = 1;
constexpr int TOTAL_ELEMENTS = N * C * H * W;

__global__ void scatter(const float *__restrict__ input,
                               const int *__restrict__ indices,
                               float *__restrict__ output) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= TOTAL_ELEMENTS) return;

  // Decode thread index
  int n = tid / (C * H * W);
  int rem = tid % (C * H * W);
  int c_idx = rem / (H * W);
  // H and W are 1, so h and w are always 0
  int h = 0;
  int w = 0;

  // Get target index from indices tensor for axis=1 (channel)
  int target_c = indices[tid];
  
  // Bounds check
  if (target_c >= 0 && target_c < 768) {
    // Calculate output index: scatter input[n][c_idx][h][w] -> output[n][target_c][h][w]
    int output_idx = n * 768 * H * W + target_c * H * W + h * W + w;
    output[output_idx] = input[tid];
  }
}

extern "C" void scatter_kernel(const float *h_input, const int *h_indices,
                              float *h_output) {
  size_t input_bytes = TOTAL_ELEMENTS * sizeof(float);
  size_t indices_bytes = TOTAL_ELEMENTS * sizeof(int);
  size_t output_bytes = N * 768 * H * W * sizeof(float); // output has shape [8,768,1,1]

  float *d_input;
  int *d_indices;
  float *d_output;

  hipMalloc(&d_input, input_bytes);
  hipMalloc(&d_indices, indices_bytes);
  hipMalloc(&d_output, output_bytes);

  // Copy input data (acts as base for output)
  hipMemcpy(d_input, h_input, input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, indices_bytes, hipMemcpyHostToDevice);

  // Initialize output with input values
  hipMemcpy(d_output, d_input, input_bytes, hipMemcpyDeviceToDevice);

  const int block_size = 256;
  int total_threads = TOTAL_ELEMENTS;
  int grid_size = (total_threads + block_size - 1) / block_size;

  scatter<<<grid_size, block_size>>>(d_input, d_indices, d_output);

  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_indices);
  hipFree(d_output);
}
