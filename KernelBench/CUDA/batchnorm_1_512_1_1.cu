
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
batchnorm(const float *__restrict__ input, float *__restrict__ output, 
          const float *__restrict__ mean, const float *__restrict__ variance,
          const float *__restrict__ gamma, const float *__restrict__ beta) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 512) return;  // total elements = 1*512*1*1 = 512

    // For input shape [1, 512, 1, 1], the index `idx` directly corresponds to channel
    // Because H=1, W=1, so each channel has only one element
    int channel_idx = idx;  // since each channel has one element, idx == channel_idx

    // Apply Batch Normalization
    float eps = 1e-5f;
    float std = sqrtf(variance[channel_idx] + eps);
    output[idx] = gamma[channel_idx] * (input[idx] - mean[channel_idx]) / std + beta[channel_idx];
}

extern "C" void batchnorm_kernel(const float *h_input, float *h_output,
                                 const float *h_mean, const float *h_variance,
                                 const float *h_gamma, const float *h_beta) {
    float *d_input, *d_output, *d_mean, *d_variance, *d_gamma, *d_beta;

    const int input_size = 512;     // 1 * 512 * 1 * 1
    const int num_channels = 512;   // C = 512

    // Allocate device memory
    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));
    hipMalloc(&d_mean, num_channels * sizeof(float));
    hipMalloc(&d_variance, num_channels * sizeof(float));
    hipMalloc(&d_gamma, num_channels * sizeof(float));
    hipMalloc(&d_beta, num_channels * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mean, h_mean, num_channels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_variance, h_variance, num_channels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gamma, h_gamma, num_channels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, num_channels * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(256);
    dim3 numBlocks((input_size + 255) / 256);  // (512 + 255) / 256 = 2 blocks

    batchnorm<<<numBlocks, blockSize>>>(d_input, d_output, d_mean, d_variance, d_gamma, d_beta);
    // Copy result back
    hipMemcpy(h_output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mean);
    hipFree(d_variance);
    hipFree(d_gamma);
    hipFree(d_beta);
}
