
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
    batchnorm(const float *__restrict__ input, float *__restrict__ output,
              const float *__restrict__ mean,
              const float *__restrict__ variance,
              const float *__restrict__ gamma, const float *__restrict__ beta) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= 512)
    return;

  int channel_idx = idx;

  float eps = 1e-5f;
  float std = sqrtf(variance[channel_idx] + eps);
  output[idx] = gamma[channel_idx] * (input[idx] - mean[channel_idx]) / std +
                beta[channel_idx];
}

extern "C" void batchnorm_kernel(const float *h_input, float *h_output,
                                 const float *h_mean, const float *h_variance,
                                 const float *h_gamma, const float *h_beta) {
  float *d_input, *d_output, *d_mean, *d_variance, *d_gamma, *d_beta;

  const int input_size = 512;
  const int num_channels = 512;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, input_size * sizeof(float));
  hipMalloc(&d_mean, num_channels * sizeof(float));
  hipMalloc(&d_variance, num_channels * sizeof(float));
  hipMalloc(&d_gamma, num_channels * sizeof(float));
  hipMalloc(&d_beta, num_channels * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_mean, h_mean, num_channels * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_variance, h_variance, num_channels * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_gamma, h_gamma, num_channels * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_beta, h_beta, num_channels * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(256);
  dim3 numBlocks((input_size + 255) / 256);

  batchnorm<<<numBlocks, blockSize>>>(d_input, d_output, d_mean, d_variance,
                                      d_gamma, d_beta);

  hipMemcpy(h_output, d_output, input_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_mean);
  hipFree(d_variance);
  hipFree(d_gamma);
  hipFree(d_beta);
}
