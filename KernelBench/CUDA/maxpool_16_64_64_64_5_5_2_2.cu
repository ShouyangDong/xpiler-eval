
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(1024)
    maxpool(float *__restrict__ A, float *__restrict__ pool_max) {
  float pool_max_local[1];
  pool_max_local[0] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_max_local[0] = max(
          pool_max_local[0],
          A[(((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) /
                   225) *
                  262144) +
                 (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) %
                    450) /
                   15) *
                  8192)) +
                (rv0 * 4096)) +
               ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 30) *
                128)) +
              (rv1 * 64)) +
             (((int)threadIdx.x) & 63))]);
    }
  }
  pool_max[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] =
      pool_max_local[0];
}

extern "C" void maxpool_kernel(float *input, float *output, int batch_size,
                               int channels, int input_H, int kernel_size,
                               int stride) {
  float *d_input;
  float *d_output;
  int output_H = (input_H - kernel_size) / stride + 1;
  int input_size = batch_size * input_H * input_H * channels;
  int output_size = batch_size * output_H * output_H * channels;
  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks((output_size + blockSize.x - 1) / blockSize.x);

  maxpool<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}
