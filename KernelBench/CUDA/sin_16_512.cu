
#include <hip/hip_runtime.h>


__global__ void __launch_bounds__(1024)
    sin(const float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 8192) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel(float *h_A, float *h_C, int total_elements) {
  float *d_A, *d_C;

  hipMalloc(&d_A, total_elements * sizeof(float));
  hipMalloc(&d_C, total_elements * sizeof(float));

  hipMemcpy(d_A, h_A, total_elements * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks((total_elements + blockSize.x - 1) / blockSize.x);

  sin<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total_elements * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
