// Generated: min along last dimension for input [4x4x4x4x4x4] -> [4x4x4x4x4]
// Total input: 4096, Reduce size: 4, Output count: 1024

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
min_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 1024) return;

    float min_val = FLT_MAX;
    for (int i = 0; i < 4; i++) {
        int in_idx = out_idx * 4 + i;
        float val = input[in_idx];
        min_val = fminf(min_val, val);
    }
    output[out_idx] = min_val;
}

extern "C" void min(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 4096;
    const int output_size = 1024;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    min_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
