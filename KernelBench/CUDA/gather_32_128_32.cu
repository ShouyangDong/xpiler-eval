#include "hip/hip_runtime.h"
// gather_axis1_32_128_32.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

// ==================== 静态维度定义 ====================
constexpr int D0 = 32;     // params.shape[0]
constexpr int D1 = 128;    // params.shape[1] (axis=1)
constexpr int D2 = 32;     // params.shape[2]
constexpr int TOTAL_PARAMS = D0 * D1 * D2;

// ============================================================ //
// Device Kernel: 沿 axis=1 gather
// 每个线程处理 output 的一个元素: output[i][n][k]
// ============================================================ //
__global__ void gather_kernel(const float* params,
                              const int64_t* indices,
                              float* output,
                              int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = D0 * N * D2;
    if (tid >= total_elements) return;

    // 计算输出位置: output[i][n][k]
    int i = tid / (N * D2);
    int rem = tid % (N * D2);
    int n = rem / D2;
    int k = rem % D2;

    int64_t src_idx = indices[n];  // 取第 n 个索引

    float val = 0.0f;
    if (src_idx >= 0 && src_idx < D1) {  // 检查 axis=1 越界
        val = params[i * D1 * D2 + src_idx * D2 + k];
    }

    output[tid] = val;
}

// ============================================================ //
// extern "C" wrapper: 包含 H2D 和 D2H 拷贝
// ============================================================ //
extern "C" void gather_kernel(const float* h_params,      // host: [32, 128, 32]
                   const int64_t* h_indices,    // host: [N]
                   float* h_output,             // host: [32, N, 32]
                   int N) {

    size_t params_bytes = D0 * D1 * D2 * sizeof(float);
    size_t indices_bytes = N * sizeof(int64_t);
    size_t output_bytes = D0 * N * D2 * sizeof(float);

    float *d_params;
    int64_t *d_indices;
    float *d_output;

    hipMalloc(&d_params, params_bytes);
    hipMalloc(&d_indices, indices_bytes);
    hipMalloc(&d_output, output_bytes);

    hipMemcpy(d_params, h_params, params_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices, indices_bytes, hipMemcpyHostToDevice);

    const int block_size = 256;
    int total_threads = D0 * N * D2;
    int grid_size = (total_threads + block_size - 1) / block_size;

    gather_kernel<<<grid_size, block_size>>>(d_params, d_indices, d_output, N);

    hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

    hipFree(d_params);
    hipFree(d_indices);
    hipFree(d_output);
}