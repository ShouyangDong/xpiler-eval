
#include <hip/hip_runtime.h>
// =============================================================================
// 1. Shape: [7, 1, 6, 7] → Total: 294 elements (matches original)
// =============================================================================
__global__ void __launch_bounds__(294)
sin(float *__restrict__ A, float *__restrict__ T_sin) {
    int idx = threadIdx.x;
    if(idx < 294){
        T_sin[idx] = sinf(A[idx]);
    }
}

extern "C" void sin_kernel(float *h_A, float *h_C, int a, int b, int c, int d) {
    float *d_A, *d_C;
    const int total = a * b * c * d;

    hipMalloc(&d_A, total * sizeof(float));
    hipMalloc(&d_C, total * sizeof(float));

    hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(294);
    dim3 numBlocks(1);

    sin<<<numBlocks, blockSize>>>(d_A, d_C);

    hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_C);
}
