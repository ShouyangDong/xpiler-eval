
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(60)
    relu(float *__restrict__ A, float *__restrict__ compute) {
  compute[((int)threadIdx.x)] = max(A[((int)threadIdx.x)], 0.000000e+00f);
}

extern "C" void relu_kernel(float *A, float *C, int size) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(60);
  dim3 numBlocks((size + 60 - 1) / 60);

  relu<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
