// gather_axis0.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

// ==================== 静态维度定义 ====================
constexpr int D0 = 2048;   // params.shape[0] (axis=0)
constexpr int D1 = 64;     // params.shape[1]
constexpr int D2 = 64;     // params.shape[2]
constexpr int SLICE_SIZE = D1 * D2;  // 每个 slice 大小
constexpr int TOTAL_PARAMS = D0 * D1 * D2;

// ============================================================ //
// Device Kernel: 沿 axis=0 gather
// 每个线程处理 output 的一个元素 output[n][i][j]
// ============================================================ //
__global__ void gather(const float* params,
                              const int64_t* indices,
                              float* output,
                              int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = N * D1 * D2;
    if (tid >= total_elements) return;

    // 计算输出位置: output[n][i][j]
    int n = tid / (D1 * D2);
    int rem = tid % (D1 * D2);
    int i = rem / D2;
    int j = rem % D2;

    int64_t src_idx = indices[n];  // 取第 n 个索引

    float val = 0.0f;
    if (src_idx >= 0 && src_idx < D0) {  // 检查 axis=0 越界
        val = params[src_idx * D1 * D2 + i * D2 + j];
    }
    // 越界 → 输出 0

    output[tid] = val;
}

// ============================================================ //
// extern "C" wrapper: 接收 host 指针，管理 device 内存
// 包含 cudaMalloc, H2D, D2H, cudaFree
// ============================================================ //
extern "C" void gather_kernel(const float* h_params,      // host: [2048, 64, 64]
                   const int64_t* h_indices,    // host: [N]
                   float* h_output,             // host: [N, 64, 64]
                   int N) {                     // indices 长度

    // 1. 计算内存大小
    size_t params_bytes = D0 * D1 * D2 * sizeof(float);
    size_t indices_bytes = N * sizeof(int64_t);
    size_t output_bytes = N * D1 * D2 * sizeof(float);

    // 2. 设备指针
    float *d_params;
    int64_t *d_indices;
    float *d_output;

    // 3. 分配设备内存
    hipMalloc(&d_params, params_bytes);
    hipMalloc(&d_indices, indices_bytes);
    hipMalloc(&d_output, output_bytes);

    // 4. H2D 拷贝
    hipMemcpy(d_params, h_params, params_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices, indices_bytes, hipMemcpyHostToDevice);

    // 5. 启动 kernel
    const int block_size = 256;
    int total_threads = N * D1 * D2;
    int grid_size = (total_threads + block_size - 1) / block_size;

    gather<<<grid_size, block_size>>>(d_params, d_indices, d_output, N);
    // 7. D2H 拷贝结果
    hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

    // 8. 释放设备内存
    hipFree(d_params);
    hipFree(d_indices);
    hipFree(d_output);
}
