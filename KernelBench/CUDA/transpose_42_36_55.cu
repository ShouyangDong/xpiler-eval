
#include <hip/hip_runtime.h>
__global__ void transpose(const float *__restrict__ input,
                          float *__restrict__ output) {

  int i0 = blockIdx.x * blockDim.x + threadIdx.x;
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int i2 = blockIdx.z * blockDim.z + threadIdx.z;
  const int d0 = 42;
  const int d1 = 36;
  const int d2 = 55;

  if (i0 < d0 && i1 < d1 && i2 < d2) {

    int in_idx = i0 * (d1 * d2) + i1 * d2 + i2;
    int out_idx = i1 * (d0 * d2) + i0 * d2 + i2;

    output[out_idx] = input[in_idx];
  }
}

extern "C" void transpose_kernel(float *host_input, float *host_output, int d0,
                                 int d1, int d2) {

  size_t in_bytes = static_cast<size_t>(d0) * d1 * d2 * sizeof(float);
  size_t out_bytes = static_cast<size_t>(d1) * d0 * d2 * sizeof(float);

  float *d_input = nullptr;
  float *d_output = nullptr;
  hipMalloc(&d_input, in_bytes);
  hipMalloc(&d_output, out_bytes);

  hipMemcpy(d_input, host_input, in_bytes, hipMemcpyHostToDevice);

  dim3 block(8, 8, 8);
  dim3 grid((d0 + block.x - 1) / block.x, (d1 + block.y - 1) / block.y,
            (d2 + block.z - 1) / block.z);

  transpose<<<grid, block>>>(d_input, d_output);
  hipDeviceSynchronize();

  hipMemcpy(host_output, d_output, out_bytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}