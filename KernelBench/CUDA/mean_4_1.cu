
#include <hip/hip_runtime.h>


__global__ void mean_kernel_dev(const float *__restrict__ input,
                                float *__restrict__ output) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= 4)
    return;

  int idx = row * 1 + 0;
  output[row] = input[idx];
}

extern "C" void mean_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 4 * 1;
  const int output_size = 4;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(4);
  dim3 numBlocks(1);

  mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}
