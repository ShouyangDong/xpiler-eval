
#include <hip/hip_runtime.h>

constexpr int D0 = 50;
constexpr int D1 = 128;
constexpr int D2 = 4;
constexpr int TOTAL_PARAMS = D0 * D1 * D2;

__global__ void gather(const float *params, const int64_t *indices,
                       float *output, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = D0 * N * D2;
  if (tid >= total_elements)
    return;

  int i = tid / (N * D2);
  int rem = tid % (N * D2);
  int n = rem / D2;
  int k = rem % D2;

  int64_t src_idx = indices[n];

  float val = 0.0f;
  if (src_idx >= 0 && src_idx < D1) {
    val = params[i * D1 * D2 + src_idx * D2 + k];
  }

  output[tid] = val;
}

extern "C" void gather_kernel(const float *h_params, const int64_t *h_indices,
                              float *h_output, int N) {

  size_t params_bytes = D0 * D1 * D2 * sizeof(float);
  size_t indices_bytes = N * sizeof(int64_t);
  size_t output_bytes = D0 * N * D2 * sizeof(float);

  float *d_params;
  int64_t *d_indices;
  float *d_output;

  hipMalloc(&d_params, params_bytes);
  hipMalloc(&d_indices, indices_bytes);
  hipMalloc(&d_output, output_bytes);

  hipMemcpy(d_params, h_params, params_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, indices_bytes, hipMemcpyHostToDevice);

  const int block_size = 256;
  int total_threads = D0 * N * D2;
  int grid_size = (total_threads + block_size - 1) / block_size;

  gather<<<grid_size, block_size>>>(d_params, d_indices, d_output, N);

  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

  hipFree(d_params);
  hipFree(d_indices);
  hipFree(d_output);
}
