
#include <hip/hip_runtime.h>


__global__ void mean_kernel_dev(const float *__restrict__ input,
                                float *__restrict__ output) {
  int tid = threadIdx.x;
  if (tid != 0)
    return;

  float sum = 0.0f;
  for (int col = 0; col < 64; col++) {
    int idx = 0 * 64 + col;
    sum += input[idx];
  }
  output[0] = sum / 64.0f;
}

extern "C" void mean_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 1 * 64;
  const int output_size = 1;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(1);
  dim3 numBlocks(1);

  mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);
  hipFree(d_input);
  hipFree(d_output);
}