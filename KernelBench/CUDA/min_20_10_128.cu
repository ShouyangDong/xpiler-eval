// Generated: min along last dimension for input [20x10x128] -> [20x10]
// Total input: 25600, Reduce size: 128, Output count: 200

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
min(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 200) return;

    float min_val = FLT_MAX;
    for (int i = 0; i < 128; i++) {
        int in_idx = out_idx * 128 + i;
        float val = input[in_idx];
        min_val = fminf(min_val, val);
    }
    output[out_idx] = min_val;
}

extern "C" void min_kernel(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 25600;
    const int output_size = 200;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    min<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
