
#include <hip/hip_runtime.h>
// =============================================================================
// CUDA Kernel for shape [64, 64, 64] → Total: 262,144 elements
// =============================================================================

__global__ void __launch_bounds__(256)
sin(const float *__restrict__ A, float *__restrict__ T_sin) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 262144) {
        T_sin[idx] = sinf(A[idx]);
    }
}

extern "C" void sin_kernel(float *h_A, float *h_C, int d0, int d1, int d2) {
    float *d_A, *d_C;
    const int total_elements = d0 * d1 * d2;  // 64*64*64 = 262144

    hipMalloc(&d_A, total_elements * sizeof(float));
    hipMalloc(&d_C, total_elements * sizeof(float));

    hipMemcpy(d_A, h_A, total_elements * sizeof(float), hipMemcpyHostToDevice);

    // Block size: 256 threads per block
    dim3 blockSize(256);
    // Grid size: ceil(total / block_size)
    dim3 numBlocks((total_elements + blockSize.x - 1) / blockSize.x);

    // Launch kernel
    sin<<<numBlocks, blockSize>>>(d_A, d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, total_elements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}