// Generated: mean along last dimension for input [1x1x1x64x64] -> [1x1x1x64]
// Total input: 4096, Reduce size: 64, Output count: 64

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
mean_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 64) return;

    float sum = 0.0f;
    for (int i = 0; i < 64; i++) {
        int in_idx = out_idx * 64 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum / 64;  // mean = sum / N
}

extern "C" void mean_kernel_1_1_1_64_64(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 4096;
    const int output_size = 64;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    mean_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
