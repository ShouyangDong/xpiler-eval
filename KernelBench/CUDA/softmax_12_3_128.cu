
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(36)
    softmax(float *__restrict__ A, float *__restrict__ T_softmax_exp) {
  if (threadIdx.x < 36) {

    float maxVal = A[threadIdx.x * 128];
    for (int i = 1; i < 128; ++i) {
      if (A[threadIdx.x * 128 + i] > maxVal) {
        maxVal = A[threadIdx.x * 128 + i];
      }
    }

    float denom = 0.0f;
    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[threadIdx.x * 128 + i] =
          expf(A[threadIdx.x * 128 + i] - maxVal);
      denom += T_softmax_exp[threadIdx.x * 128 + i];
    }

    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[threadIdx.x * 128 + i] /= denom;
    }
  }
}

extern "C" void softmax_kernel(float *A, float *C, int size1, int size2) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size1 * size2 * sizeof(float));
  hipMalloc(&d_C, size1 * size2 * sizeof(float));

  hipMemcpy(d_A, A, size1 * size2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(36);
  dim3 numBlocks((size1 + 36 - 1) / 36);

  softmax<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size1 * size2 * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
