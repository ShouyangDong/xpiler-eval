
#include <hip/hip_runtime.h>

__global__ void transpose(const float *__restrict__ input,
                          float *__restrict__ output) {
  int i0 = blockIdx.z;
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  const int d0 = 33;
  const int d1 = 40;
  const int d2 = 5;
  if (i0 < d0 && i1 < d1 && i2 < d2) {
    int in_idx = i0 * (d1 * d2) + i1 * d2 + i2;
    int out_idx = i0 * (d2 * d1) + i2 * d1 + i1;
    output[out_idx] = input[in_idx];
  }
}

extern "C" void transpose_kernel(float *input, float *output, int d0, int d1,
                                 int d2) {

  size_t total_elems = d0 * d1 * d2;
  size_t bytes = total_elems * sizeof(float);

  float *d_input, *d_output;
  hipMalloc(&d_input, bytes);
  hipMalloc(&d_output, bytes);

  hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

  dim3 block(16, 16);
  dim3 grid((d2 + block.x - 1) / block.x, (d1 + block.y - 1) / block.y, d0);

  transpose<<<grid, block>>>(d_input, d_output);

  hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}
