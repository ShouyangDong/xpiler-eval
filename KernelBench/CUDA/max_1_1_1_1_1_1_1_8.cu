// Generated: max along last dimension for input [1x1x1x1x1x1x1x8] -> [1x1x1x1x1x1x1]
// Total input: 8, Reduce size: 8, Output count: 1

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
max_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 1) return;

    float max_val = -FLT_MAX;
    for (int i = 0; i < 8; i++) {
        int in_idx = out_idx * 8 + i;
        float val = input[in_idx];
        max_val = fmaxf(max_val, val);
    }
    output[out_idx] = max_val;
}

extern "C" void max(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 8;
    const int output_size = 1;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    max_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
