
#include <hip/hip_runtime.h>
// =============================================================================
// 2. Shape: [1, 3, 224, 224] → Total: 150,528 elements (Image input)
// =============================================================================
__global__ void __launch_bounds__(1024)
sub(float *__restrict__ A, float *__restrict__ B, float *__restrict__ C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 150528) {
        C[idx] = A[idx] - B[idx];
    }
}

extern "C" void sub_kernel(float *h_A, float *h_B, float *h_C) {
    float *d_A, *d_B, *d_C;
    const int total = 1 * 3 * 224 * 224;  // 150528

    // 分配设备内存
    hipMalloc(&d_A, total * sizeof(float));
    hipMalloc(&d_B, total * sizeof(float));
    hipMalloc(&d_C, total * sizeof(float));

    // Host → Device
    hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total * sizeof(float), hipMemcpyHostToDevice);

    // 配置 kernel 启动参数
    dim3 blockSize(1024);
    dim3 numBlocks((total + blockSize.x - 1) / blockSize.x);  // 上取整

    // ✅ 修复：使用正确的 kernel 名字 'sub'
    sub<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    // 等待 kernel 完成（可选，但建议用于调试）
    hipDeviceSynchronize();

    // Device → Host
    hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
