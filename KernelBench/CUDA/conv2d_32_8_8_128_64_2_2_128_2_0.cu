
#include <hip/hip_runtime.h>
__global__ void conv2d(float *input, float *kernel, float *output) {
  int bs = blockIdx.z;
  int oc = threadIdx.x;
  int oh = blockIdx.y;
  int ow = blockIdx.x;

  if (oc < 64 && oh < 4 && ow < 4 && bs < 32) {
    float sum = 0.0;

    for (int kh = 0; kh < 2; kh++) {
      for (int kw = 0; kw < 2; kw++) {
        for (int ic = 0; ic < 128; ic++) {
          int ih = oh * 2 + kh;
          int iw = ow * 2 + kw;

          int input_idx = bs * (8 * 8 * 128) + ih * (8 * 128) + iw * 128 + ic;

          int kernel_idx = oc * (2 * 2 * 128) + kh * (2 * 128) + kw * 128 + ic;

          sum += input[input_idx] * kernel[kernel_idx];
        }
      }
    }

    int output_idx = bs * (4 * 4 * 64) + oh * (4 * 64) + ow * 64 + oc;

    output[output_idx] = sum;
  }
}

extern "C" void conv2d_kernel(float *input, float *filter, float *output,
                              int batch_size, int input_height,
                              int input_channels, int output_channels,
                              int kernel_height, int stride) {
  int output_height = (input_height - kernel_height) / stride + 1;
  int output_width = (input_height - kernel_height) / stride + 1;

  int input_size = batch_size * input_height * input_height * input_channels;
  int kernel_size =
      output_channels * kernel_height * kernel_height * input_channels;
  int output_size = batch_size * output_height * output_width * output_channels;

  float *d_input, *d_filter, *d_output;
  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_filter, kernel_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, input, input_size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_filter, filter, kernel_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(output_channels);
  dim3 numBlocks(output_width, output_height, batch_size);

  conv2d<<<numBlocks, blockSize>>>(d_input, d_filter, d_output);

  hipMemcpy(output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_filter);
  hipFree(d_output);
}
