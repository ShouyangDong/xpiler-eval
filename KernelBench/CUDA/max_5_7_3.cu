// Generated: max along last dimension for input [5x7x3] -> [5x7]
// Total input: 105, Reduce size: 3, Output count: 35

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
max(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 35) return;

    float max_val = -FLT_MAX;
    for (int i = 0; i < 3; i++) {
        int in_idx = out_idx * 3 + i;
        float val = input[in_idx];
        max_val = fmaxf(max_val, val);
    }
    output[out_idx] = max_val;
}

extern "C" void max_kernel(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 105;
    const int output_size = 35;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    max<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
