
#include <hip/hip_runtime.h>
// ============================================================ //
// 实例 4: (100, 32), indices=16
// ============================================================ //

__global__ void gather(const float* params,
                                     const int* indices,
                                     float* output) {
  constexpr int PARAMS_BATCH = 100;
  constexpr int PARAMS_LEN   = 32;
  constexpr int INDICES_LEN  = 16;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= INDICES_LEN) return;

  int idx = indices[i];
  float* out_row = &output[i * PARAMS_LEN];

  if (idx < 0 || idx >= PARAMS_BATCH) {
    // 越界：置零
    for (int j = 0; j < PARAMS_LEN; ++j) {
      out_row[j] = 0.0f;
    }
  } else {
    const float* src_row = &params[idx * PARAMS_LEN];
    for (int j = 0; j < PARAMS_LEN; ++j) {
      out_row[j] = src_row[j];
    }
  }
}


extern "C" void gather_kernl(const float* d_params,
                                     const int* d_indices,
                                     float* d_output,
                                     int size1,
                                     int size2,
                                     int size3) {

  float *d_A;
  float *d_B;
  float *d_C;

  hipMalloc(&d_A, size1 * sizeof(float));
  hipMalloc(&d_B, size2 * sizeof(int));
  hipMalloc(&d_C, size3 * sizeof(float));

  hipMemcpy(d_A, d_params, size1 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, d_indices, size2 * sizeof(int), hipMemcpyHostToDevice);

  constexpr int block_size = 16;
  constexpr int grid_size  = 1;

  gather<<<grid_size, block_size>>>(d_params, d_indices, d_output);

  hipMemcpy(d_output, d_C, size3 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}