
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
copy_reshape(const float *__restrict__ input, float *__restrict__ output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int total = 512;

    for (int i = idx; i < total; i += stride) {
        output[i] = input[i];
    }
}

extern "C" void reshape_kernel_8_8_8_to_512_1_1(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int total_elements = 512;

    hipMalloc(&d_input, total_elements * sizeof(float));
    hipMalloc(&d_output, total_elements * sizeof(float));

    hipMemcpy(d_input, h_input, total_elements * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((total_elements + 255) / 256);

    copy_reshape<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, total_elements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
