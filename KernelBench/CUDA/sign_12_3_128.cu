
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(1024)
    sign(float *__restrict__ A, float *__restrict__ T_sign) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 4608) {
    T_sign[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] =
        ((0.000000e+00f < A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))])
             ? 1.000000e+00f
             : ((A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] <
                 0.000000e+00f)
                    ? -1.000000e+00f
                    : 0.000000e+00f));
  }
}

extern "C" void sign_kernel(float *A, float *C, int size) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks((size + 1024 - 1) / 1024);

  sign<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
