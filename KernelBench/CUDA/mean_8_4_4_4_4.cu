// Generated: mean along last dimension for input [8x4x4x4x4] -> [8x4x4x4]
// Total input: 2048, Reduce size: 4, Output count: 512

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
mean_last_dim(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 512) return;

    float sum = 0.0f;
    for (int i = 0; i < 4; i++) {
        int in_idx = out_idx * 4 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum / 4;  // mean = sum / N
}

extern "C" void mean_kernel_8_4_4_4_4(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 2048;
    const int output_size = 512;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    mean_last_dim<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
