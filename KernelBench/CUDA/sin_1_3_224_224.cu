
#include <hip/hip_runtime.h>


__global__ void __launch_bounds__(1024)
    sin(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 150528) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel(float *h_A, float *h_C, int n, int h, int w, int c) {
  float *d_A, *d_C;
  const int total = n * w * h * c;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((total + 293) / 294);

  sin<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_32x64(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 2048) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel_32x64(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 32 * 64;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((total + 293) / 294);

  sin_32x64<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_8x16x32x32(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 131072) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel_8x16x32x32(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 8 * 16 * 32 * 32;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((total + 293) / 294);

  sin_8x16x32x32<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_1x512(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = threadIdx.x;
  if (idx < 512) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel_1x512(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 512;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks(1);

  sin_1x512<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_64x64x64(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 262144) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel_64x64x64(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 64 * 64 * 64;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((total + 293) / 294);

  sin_64x64x64<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_2x1x1024(float *__restrict__ A, float *__restrict__ T_sin) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 2048) {
    T_sin[idx] = sinf(A[idx]);
  }
}

extern "C" void sin_kernel_2x1x1024(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 2 * 1 * 1024;

  hipMalloc(&d_A, total * sizeof(float));
  hipMalloc(&d_C, total * sizeof(float));

  hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((total + 293) / 294);

  sin_2x1x1024<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}

__global__ void __launch_bounds__(294)
    sin_scalar(float *__restrict__ A, float *__restrict__ T_sin) {
  if (threadIdx.x == 0) {
    T_sin[0] = sinf(A[0]);
  }
}

extern "C" void sin_kernel_scalar(float *h_A, float *h_C) {
  float *d_A, *d_C;
  const int total = 1;

  hipMalloc(&d_A, sizeof(float));
  hipMalloc(&d_C, sizeof(float));

  hipMemcpy(d_A, h_A, sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks(1);

  sin_scalar<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(h_C, d_C, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}