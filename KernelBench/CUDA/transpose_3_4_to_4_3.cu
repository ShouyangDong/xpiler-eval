// Generated: Transpose from [3x4] to [4x3]
// Axes: (1 0), Total elements: 12

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
transpose_kernel(const float *__restrict__ input, float *__restrict__ output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 12) return;

    // Step 1: Flatten index -> multi-dimensional indices (input shape)
    int in_indices[2];
    int tmp = idx;
    in_indices[1] = tmp % 4;
    tmp /= 4;
    in_indices[0] = tmp % 3;
    tmp /= 3;

    // Step 2: Permute indices according to axes
    int out_indices[2];
    out_indices[0] = in_indices[1];
    out_indices[1] = in_indices[0];

    // Step 3: Multi-dimensional indices -> linear index (output shape)
    int out_idx = 0;
    out_idx += out_indices[1];
    out_idx += out_indices[0] * 3;

    // Write to output
    output[out_idx] = input[idx];
}

extern "C" void transpose_kernel_3_4_to_4_3(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int total = 12;

    hipMalloc(&d_input, total * sizeof(float));
    hipMalloc(&d_output, total * sizeof(float));

    hipMemcpy(d_input, h_input, total * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((total + 255) / 256);

    transpose_kernel<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, total * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
