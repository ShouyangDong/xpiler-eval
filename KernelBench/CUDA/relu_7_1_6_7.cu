
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(294)
    relu(float *__restrict__ A, float *__restrict__ compute) {
  compute[((int)threadIdx.x)] = max(A[((int)threadIdx.x)], 0.000000e+00f);
}

extern "C" void relu_kernel(float *A, float *C, int size) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(294);
  dim3 numBlocks((size + 294 - 1) / 294);

  relu<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
