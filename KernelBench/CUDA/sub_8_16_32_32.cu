
#include <hip/hip_runtime.h>
// =============================================================================
// 4. Shape: [8, 16, 32, 32] → Total: 131,072 elements (CNN intermediate)
// =============================================================================
__global__ void __launch_bounds__(960)
sub_8x16x32x32(float *__restrict__ A, float *__restrict__ B, float *__restrict__ C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 131072) {
        C[idx] = A[idx] - B[idx];
    }
}

extern "C" void sub_kernel_8x16x32x32(float *h_A, float *h_B, float *h_C) {
    float *d_A, *d_B, *d_C;
    const int total = 8 * 16 * 32 * 32;

    hipMalloc(&d_A, total * sizeof(float));
    hipMalloc(&d_B, total * sizeof(float));
    hipMalloc(&d_C, total * sizeof(float));

    hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(960);
    dim3 numBlocks((total + 959) / 960);

    sub_8x16x32x32<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}