// Generated: min along last dimension for input [5x5x5x10] -> [5x5x5]
// Total input: 1250, Reduce size: 10, Output count: 125

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void __launch_bounds__(256)
min(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 125) return;

    float min_val = FLT_MAX;
    for (int i = 0; i < 10; i++) {
        int in_idx = out_idx * 10 + i;
        float val = input[in_idx];
        min_val = fminf(min_val, val);
    }
    output[out_idx] = min_val;
}

extern "C" void min_kernel(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 1250;
    const int output_size = 125;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    min<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
