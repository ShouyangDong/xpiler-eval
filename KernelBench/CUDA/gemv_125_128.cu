
#include <hip/hip_runtime.h>
__global__ void gemv(float *A, float *x, float *y) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < 125) {
    float sum = 0.0f;
    for (int i = 0; i < 128; i++) {
      sum += A[row * 128 + i] * x[i];
    }
    y[row] = sum;
  }
}

extern "C" void gemv_kernel(float *A, float *x, float *y, int m, int n) {
  float *d_A, *d_x, *d_y;

  hipMalloc(&d_A, m * n * sizeof(float));
  hipMalloc(&d_x, n * sizeof(float));
  hipMalloc(&d_y, m * sizeof(float));

  hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 125;
  int numBlocks = (m + blockSize - 1) / blockSize;

  gemv<<<numBlocks, blockSize>>>(d_A, d_x, d_y);

  hipMemcpy(y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_y);
}
