
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
sum(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 1) return;

    float sum = 0.0f;
    for (int i = 0; i < 512; i++) {
        int in_idx = out_idx * 512 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum;
}

extern "C" void sum_kernel(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 512;
    const int output_size = 1;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    sum<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
