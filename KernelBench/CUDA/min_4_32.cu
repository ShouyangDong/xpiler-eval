#include "hip/hip_runtime.h"
// Kernel: reduce along axis=1 for input [4, 32] -> output [4]
// Each thread handles one row
__global__ void min_kernel_dev(const float* __restrict__ input, float* __restrict__ output) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= 4) return;  // Only 4 rows

    float min_val = FLT_MAX;  // initialize to +inf
    for (int col = 0; col < 32; col++) {
        int idx = row * 32 + col;  // input[row][col]
        min_val = fminf(min_val, input[idx]);
    }
    output[row] = min_val;
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C" void min_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 4 * 32;   // 128
        const int output_size = 4;       // 4

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(4);
        dim3 numBlocks(1);  // 4 threads → one block is enough

        min_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

        // Copy result back to host
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_input);
        hipFree(d_output);
}