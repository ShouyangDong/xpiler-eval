// Generated: Transpose from [2x3x4x5x6] to [3x2x6x4x5]
// Axes: (1 0 4 2 3), Total elements: 720

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
transpose_kernel(const float *__restrict__ input, float *__restrict__ output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 720) return;

    // Step 1: Flatten index -> multi-dimensional indices (input shape)
    int in_indices[5];
    int tmp = idx;
    in_indices[4] = tmp % 6;
    tmp /= 6;
    in_indices[3] = tmp % 5;
    tmp /= 5;
    in_indices[2] = tmp % 4;
    tmp /= 4;
    in_indices[1] = tmp % 3;
    tmp /= 3;
    in_indices[0] = tmp % 2;
    tmp /= 2;

    // Step 2: Permute indices according to axes
    int out_indices[5];
    out_indices[0] = in_indices[1];
    out_indices[1] = in_indices[0];
    out_indices[2] = in_indices[4];
    out_indices[3] = in_indices[2];
    out_indices[4] = in_indices[3];

    // Step 3: Multi-dimensional indices -> linear index (output shape)
    int out_idx = 0;
    out_idx += out_indices[4];
    out_idx += out_indices[3] * 5;
    out_idx += out_indices[2] * 20;
    out_idx += out_indices[1] * 120;
    out_idx += out_indices[0] * 240;

    // Write to output
    output[out_idx] = input[idx];
}

extern "C" void transpose_kernel_2_3_4_5_6_to_3_2_6_4_5(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int total = 720;

    hipMalloc(&d_input, total * sizeof(float));
    hipMalloc(&d_output, total * sizeof(float));

    hipMemcpy(d_input, h_input, total * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((total + 255) / 256);

    transpose_kernel<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, total * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
