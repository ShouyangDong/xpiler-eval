
#include <hip/hip_runtime.h>


constexpr int N = 4;
constexpr int C = 32;
constexpr int H = 112;
constexpr int W = 112;
constexpr int TOTAL_ELEMENTS = N * C * H * W;
constexpr int OUTPUT_C =
    C * 2; // Concatenating two tensors of 32 channels -> 64 channels
constexpr int OUTPUT_TOTAL_ELEMENTS = N * OUTPUT_C * H * W;

__global__ void concat(const float *__restrict__ input1,
                       const float *__restrict__ input2,
                       float *__restrict__ output) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= OUTPUT_TOTAL_ELEMENTS)
    return;

  // Decode output index
  int n = tid / (OUTPUT_C * H * W);
  int rem = tid % (OUTPUT_C * H * W);
  int c = rem / (H * W);
  rem = rem % (H * W);
  int h = rem / W;
  int w = rem % W;

  if (c < C) {
    // First half comes from input1
    output[tid] = input1[n * C * H * W + c * H * W + h * W + w];
  } else {
    // Second half comes from input2
    int c2 = c - C;
    output[tid] = input2[n * C * H * W + c2 * H * W + h * W + w];
  }
}

extern "C" void concat_kernel(const float *h_input1, const float *h_input2,
                              float *h_output) {
  size_t input_bytes = TOTAL_ELEMENTS * sizeof(float);
  size_t output_bytes = OUTPUT_TOTAL_ELEMENTS * sizeof(float);

  float *d_input1;
  float *d_input2;
  float *d_output;

  hipMalloc(&d_input1, input_bytes);
  hipMalloc(&d_input2, input_bytes);
  hipMalloc(&d_output, output_bytes);

  hipMemcpy(d_input1, h_input1, input_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_input2, h_input2, input_bytes, hipMemcpyHostToDevice);

  const int block_size = 256;
  int total_threads = OUTPUT_TOTAL_ELEMENTS;
  int grid_size = (total_threads + block_size - 1) / block_size;

  concat<<<grid_size, block_size>>>(d_input1, d_input2, d_output);

  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

  hipFree(d_input1);
  hipFree(d_input2);
  hipFree(d_output);
}
