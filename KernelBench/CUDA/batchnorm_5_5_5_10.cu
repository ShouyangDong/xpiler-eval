
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
batchnorm(const float *__restrict__ input, float *__restrict__ output, 
          const float *__restrict__ mean, const float *__restrict__ variance,
          const float *__restrict__ gamma, const float *__restrict__ beta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 1250) return;

    // Calculate the offset of the current element within its feature map
    int tmp_idx = idx;
    int offsets[4];
    for (int i = 0; i < 4; ++i) {
        offsets[4-i-1] = tmp_idx % 10;
        tmp_idx /= 10;
    }

    // Calculate the index for the channel (assuming channels are the second dimension)
    int channel_idx = offsets[1];

    // Apply Batch Normalization formula
    output[idx] = gamma[channel_idx] * (input[idx] - mean[channel_idx]) / sqrt(variance[channel_idx] + 1e-5f) + beta[channel_idx];
}

extern "C" void batchnorm_kernel_5_5_5_10(const float *h_input, float *h_output,
                                                       const float *h_mean, const float *h_variance,
                                                       const float *h_gamma, const float *h_beta) {
    float *d_input, *d_output, *d_mean, *d_variance, *d_gamma, *d_beta;
    const int input_size = 1250;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));
    hipMalloc(&d_mean, 5 * sizeof(float));
    hipMalloc(&d_variance, 5 * sizeof(float));
    hipMalloc(&d_gamma, 5 * sizeof(float));
    hipMalloc(&d_beta, 5 * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mean, h_mean, 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_variance, h_variance, 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gamma, h_gamma, 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, 5 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((input_size + 255) / 256);

    batchnorm<<<numBlocks, blockSize>>>(d_input, d_output, d_mean, d_variance, d_gamma, d_beta);

    hipMemcpy(h_output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mean);
    hipFree(d_variance);
    hipFree(d_gamma);
    hipFree(d_beta);
}
