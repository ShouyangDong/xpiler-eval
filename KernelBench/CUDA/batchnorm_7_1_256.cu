
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(256)
batchnorm(const float *__restrict__ input, float *__restrict__ output, 
          const float *__restrict__ mean, const float *__restrict__ variance,
          const float *__restrict__ gamma, const float *__restrict__ beta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 1792) return;

    // Calculate the offset of the current element within its feature map
    int tmp_idx = idx;
    int offsets[3];
    for (int i = 0; i < 3; ++i) {
        offsets[3-i-1] = tmp_idx % 256;
        tmp_idx /= 256;
    }

    // Calculate the index for the channel (assuming channels are the second dimension)
    int channel_idx = offsets[1];

    // Apply Batch Normalization formula
    output[idx] = gamma[channel_idx] * (input[idx] - mean[channel_idx]) / sqrt(variance[channel_idx] + 1e-5f) + beta[channel_idx];
}

extern "C" void batchnorm_kernel_7_1_256(const float *h_input, float *h_output,
                                                       const float *h_mean, const float *h_variance,
                                                       const float *h_gamma, const float *h_beta) {
    float *d_input, *d_output, *d_mean, *d_variance, *d_gamma, *d_beta;
    const int input_size = 1792;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));
    hipMalloc(&d_mean, 7 * sizeof(float));
    hipMalloc(&d_variance, 7 * sizeof(float));
    hipMalloc(&d_gamma, 7 * sizeof(float));
    hipMalloc(&d_beta, 7 * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mean, h_mean, 7 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_variance, h_variance, 7 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gamma, h_gamma, 7 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, 7 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((input_size + 255) / 256);

    batchnorm<<<numBlocks, blockSize>>>(d_input, d_output, d_mean, d_variance, d_gamma, d_beta);

    hipMemcpy(h_output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mean);
    hipFree(d_variance);
    hipFree(d_gamma);
    hipFree(d_beta);
}
