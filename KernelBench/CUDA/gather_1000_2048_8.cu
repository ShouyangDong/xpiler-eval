#include <hip/hip_runtime.h>
#include <stdio.h>

// 固定参数（根据你的说明）
constexpr int DIM0        = 1000;  // 第一维大小
constexpr int DIM1        = 2048;  // 第二维大小
constexpr int PARAMS_DIM2 = 8;     // 第三维大小（你明确说“输入就是1000,2048,8”）
constexpr int INDICES_LEN = 8;     // indices 长度（来自 args[2]）

// ============================================================ //
// 核函数：gather 沿 axis=2
// output[i][j][k] = params[i][j][ indices[k] ]
// 每个线程处理一个 (i, j, k) 输出元素
// ============================================================ //
__global__ void gather(const float* params,
                       const int* indices,
                       float* output) {
  int k = threadIdx.x;                    // indices 维度 [0, 7]
  int j = blockIdx.x * blockDim.x + k;    // DIM1 维度 [0, 2047]
  int i = blockIdx.y;                     // DIM0 维度 [0, 999]

  if (i >= DIM0 || j >= DIM1 || k >= INDICES_LEN) return;

  int feat_idx = indices[k];  // 要取的特征索引

  float val = 0.0f;
  if (feat_idx >= 0 && feat_idx < PARAMS_DIM2) {
    val = params[i * DIM1 * PARAMS_DIM2 + j * PARAMS_DIM2 + feat_idx];
  }

  output[i * DIM1 * INDICES_LEN + j * INDICES_LEN + k] = val;
}

// ============================================================ //
// Host 函数：包含 H2D、D2H、内存管理
// ============================================================ //
extern "C" void gather_kernel(const float* h_params,
                              const int* h_indices,
                              float* h_output) {
  float *d_params;
  int *d_indices;
  float *d_output;

  size_t params_bytes  = DIM0 * DIM1 * PARAMS_DIM2 * sizeof(float);
  size_t indices_bytes = INDICES_LEN * sizeof(int);
  size_t output_bytes  = DIM0 * DIM1 * INDICES_LEN * sizeof(float);

  // 1. 分配设备内存
  hipMalloc(&d_params, params_bytes);
  hipMalloc(&d_indices, indices_bytes);
  hipMalloc(&d_output, output_bytes);

  // 2. Host to Device 拷贝
  hipMemcpy(d_params, h_params, params_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, indices_bytes, hipMemcpyHostToDevice);

  // 3. 配置 kernel 启动参数
  dim3 block_size(INDICES_LEN, 32);  // x: 8 threads (indices), y: 32 并行处理 j
  dim3 grid_size(
    (DIM1 + 31) / 32,  // 向上取整覆盖所有 j
    DIM0               // 每个 i 一个 block_y
  );

  // 4. 启动 kernel
  gather<<<grid_size, block_size>>>(d_params, d_indices, d_output);

  // 5. 错误检查
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    goto cleanup;
  }
  hipDeviceSynchronize();

  // 6. Device to Host 拷贝结果
  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

cleanup:
  // 7. 释放设备内存
  hipFree(d_params);
  hipFree(d_indices);
  hipFree(d_output);
}