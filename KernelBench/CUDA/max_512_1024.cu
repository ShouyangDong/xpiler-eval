#include "hip/hip_runtime.h"


__global__ void max_dev(const float *__restrict__ input,
                        float *__restrict__ output) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col >= 1024)
    return;

  float max_val = -FLT_MAX;
  for (int row = 0; row < 512; row++) {
    int idx = row * 1024 + col;
    max_val = fmaxf(max_val, input[idx]);
  }
  output[col] = max_val;
}

extern "C" void max_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 512 * 1024;
  const int output_size = 1024;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(256);
  dim3 numBlocks((1024 + 255) / 256);

  max_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}
