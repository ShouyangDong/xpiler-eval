
#include <hip/hip_runtime.h>


__global__ void rmsnorm(float *A, float *B) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float eps = 1e-5f;

  if (idx < 8192) {

    float sum = 0.0;
    for (int j = 0; j < 4096; j++) {
      sum += A[idx * 4096 + j] * A[idx * 4096 + j];
    }

    float mean = sum / 4096;

    float scale = 1.0 / sqrt(mean + eps);

    for (int j = 0; j < 4096; j++) {
      B[idx * 4096 + j] = A[idx * 4096 + j] * scale;
    }
  }
}

extern "C" void rmsnorm_kernel(float *A, float *B, int size_1, int size_2) {

  float *d_A, *d_B;
  int num_elements = size_1 * size_2;
  hipMalloc(&d_A, num_elements * sizeof(float));
  hipMalloc(&d_B, num_elements * sizeof(float));

  hipMemcpy(d_A, A, num_elements * sizeof(float), hipMemcpyHostToDevice);

  int block_size = 1024;
  int num_blocks = (size_1 + block_size - 1) / block_size;

  rmsnorm<<<num_blocks, block_size>>>(d_A, d_B);

  hipMemcpy(B, d_B, num_elements * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
}
