// Generated: mean along last dimension for input [3x3x3x3x3x3] -> [3x3x3x3x3]
// Total input: 729, Reduce size: 3, Output count: 243

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
mean(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 243) return;

    float sum = 0.0f;
    for (int i = 0; i < 3; i++) {
        int in_idx = out_idx * 3 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum / 3;  // mean = sum / N
}

extern "C" void mean_kernel_3_3_3_3_3_3(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 729;
    const int output_size = 243;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    mean<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
