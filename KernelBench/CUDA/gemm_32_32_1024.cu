#include "hip/hip_runtime.h"
__global__ void gemm(half *A, half *B, float *C) {
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

  int blockRow = blockIdx.y * 16;
  int blockCol = blockIdx.x * 16;

  if (blockRow < 32 && blockCol < 1024) {

    wmma::fill_fragment(c_frag, 0.0f);

    for (int k = 0; k < 32; k += 16) {

      wmma::load_matrix_sync(a_frag, A + blockRow * 32 + k, 32);
      wmma::load_matrix_sync(b_frag, B + k * 1024 + blockCol, 1024);

      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    wmma::store_matrix_sync(C + blockRow * 1024 + blockCol, c_frag, 1024,
                            wmma::mem_row_major);
  }
}

extern "C" void gemm_kernel(half *A, half *B, float *C,  int m, int k, int n) {
  half *d_A;
  half *d_B;
  float *d_C;

  hipMalloc(&d_A, m * k * sizeof(half));
  hipMalloc(&d_B, k * n * sizeof(half));
  hipMalloc(&d_C, m * n * sizeof(float));

  hipMemcpy(d_A, A, m * k * sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, k * n * sizeof(half), hipMemcpyHostToDevice);

  dim3 blockSize(32);
  dim3 numBlocks((n + 16 - 1) / 16, (m + 16 - 1) / 16);
  gemm<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
