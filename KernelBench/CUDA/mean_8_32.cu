
#include <hip/hip_runtime.h>
// Kernel: reduce along axis=0 for input [8, 32] -> output [32]
// Each thread handles one column
__global__ void mean_kernel_dev(const float* __restrict__ input, float* __restrict__ output) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= 32) return;  // Only 32 columns

    float sum = 0.0f;
    for (int row = 0; row < 8; row++) {
        int idx = row * 32 + col;  // input[row][col]
        sum += input[idx];
    }
    output[col] = sum / 8.0f;  // Divide by reduction size
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C" void mean_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 8 * 32;   // 256
        const int output_size = 32;      // 32

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(32);
        dim3 numBlocks(1);  // 32 threads → one block is enough

        mean_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

        // Copy result back to host
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_input);
        hipFree(d_output);

}