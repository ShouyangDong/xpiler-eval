
#include <hip/hip_runtime.h>
// =============================================================================
// CUDA Kernel for shape [32, 64] → Total: 2,048 elements
// =============================================================================

__global__ void __launch_bounds__(1024)sin_dev(const float *__restrict__ A, float *__restrict__ T_sin) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 2048) {
        T_sin[idx] = sinf(A[idx]);
    }
}

extern "C" void sin_kernel(float *h_A, float *h_C, int total_elements) {
    float *d_A, *d_C;

    hipMalloc(&d_A, total_elements * sizeof(float));
    hipMalloc(&d_C, total_elements * sizeof(float));

    hipMemcpy(d_A, h_A, total_elements * sizeof(float), hipMemcpyHostToDevice);

    // 使用 256 线程每 block（常见且高效）
    dim3 blockSize(1024);
    // 计算所需 block 数量：ceil(2048 / 256) = 8
    dim3 numBlocks((total_elements + blockSize.x - 1) / blockSize.x);

    // 启动 kernel
    sin_dev<<<numBlocks, blockSize>>>(d_A, d_C);

    hipMemcpy(h_C, d_C, total_elements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}
