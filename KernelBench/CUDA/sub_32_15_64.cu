
#include <hip/hip_runtime.h>
// =============================================================================
// Kernel: sub for shape [32, 15, 64] → Total: 30,720 elements
// =============================================================================
__global__ void __launch_bounds__(1024)
sub(const float* __restrict__ A,
             const float* __restrict__ B,
             float* __restrict__ C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 30720) {
        C[idx] = A[idx] - B[idx];
    }
}

// =============================================================================
// Host wrapper function
// =============================================================================
extern "C" void sub_kernel(float* h_A, float* h_B, float* h_C) {
    float *d_A, *d_B, *d_C;
    const int total = 32 * 15 * 64;  // 30720

    // 分配 GPU 设备内存
    hipMalloc(&d_A, total * sizeof(float));
    hipMalloc(&d_B, total * sizeof(float));
    hipMalloc(&d_C, total * sizeof(float));

    // Host → Device 拷贝
    hipMemcpy(d_A, h_A, total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, total * sizeof(float), hipMemcpyHostToDevice);

    // 配置 kernel 启动参数
    dim3 blockSize(256);                    // 每 block 256 个线程
    dim3 numBlocks((total + 255) / 256);    // 上取整: (30720 + 255) / 256 = 120

    // 启动 kernel
    sub<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    // 同步并检查错误
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Device → Host 拷贝结果
    hipMemcpy(h_C, d_C, total * sizeof(float), hipMemcpyDeviceToHost);

    // 释放 GPU 内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}