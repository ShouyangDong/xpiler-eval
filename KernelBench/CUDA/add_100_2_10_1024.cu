
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(1024)
    add(float *__restrict__ A, float *__restrict__ B,
        float *__restrict__ T_add) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0;
       ax0_ax1_fused_ax2_fused_ax3_fused_outer < 8;
       ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) +
         ((int)blockIdx.x)) < 2048000) {
      T_add[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) +
              (((int)blockIdx.x) * 1024)) +
             ((int)threadIdx.x))] =
          (A[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) +
               (((int)blockIdx.x) * 1024)) +
              ((int)threadIdx.x))] +
           B[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) +
               (((int)blockIdx.x) * 1024)) +
              ((int)threadIdx.x))]);
    }
  }
}

extern "C" void add_kernel(float *A, float *B, float *C, int size) {
  float *d_A;
  float *d_B;
  float *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_B, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks(256);
  add<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
