
#include <hip/hip_runtime.h>
__global__ void rmsnorm(float *A, float *B) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float eps = 1e-5f;

  if (idx < 2048) {
    // Calculate sum
    float sum = 0.0;
    for (int j = 0; j < 2048; j++) {
      sum += A[idx * 2048 + j] * A[idx * 2048 + j];
    }

    // Calculate mean
    float mean = sum / 2048;

    // Calculate scale
    float scale = 1.0 / sqrt(mean + eps);

    // Normalize and store in B
    for (int j = 0; j < 2048; j++) {
      B[idx * 2048 + j] = A[idx * 2048 + j] * scale;
    }
  }
}

extern "C" void rmsnorm_kernel(float *A, float *B, int size_1, int size_2) {
  // Allocate memory on the device
  float *d_A, *d_B;
  int num_elements = size_1 * size_2;
  hipMalloc(&d_A, num_elements * sizeof(float));
  hipMalloc(&d_B, num_elements * sizeof(float));

  // Copy data from host to device
  hipMemcpy(d_A, A, num_elements * sizeof(float), hipMemcpyHostToDevice);

  // Define grid and block dimensions
  int block_size = 1024;
  int num_blocks = (size_1 + block_size - 1) / block_size;

  // Launch kernel
  rmsnorm<<<num_blocks, block_size>>>(d_A, d_B);
  // Copy the result back to host
  hipMemcpy(B, d_B, num_elements * sizeof(float), hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
}
