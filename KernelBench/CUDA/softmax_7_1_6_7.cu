
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(42)
    softmax(float *__restrict__ A, float *__restrict__ T_softmax_norm) {
  if (threadIdx.x < 42) {
    int rowStart = threadIdx.x * 7;

    float maxVal = A[rowStart];
    for (int i = 1; i < 7; ++i) {
      if (A[rowStart + i] > maxVal) {
        maxVal = A[rowStart + i];
      }
    }

    float denom = 0.0f;
    for (int i = 0; i < 7; ++i) {
      T_softmax_norm[rowStart + i] = expf(A[rowStart + i] - maxVal);
      denom += T_softmax_norm[rowStart + i];
    }

    for (int i = 0; i < 7; ++i) {
      T_softmax_norm[rowStart + i] /= denom;
    }
  }
}

extern "C" void softmax_kernel(float *A, float *C, int size1, int size2) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size1 * size2 * sizeof(float));
  hipMalloc(&d_C, size1 * size2 * sizeof(float));

  hipMemcpy(d_A, A, size1 * size2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(42);
  dim3 numBlocks((size1 + 42 - 1) / 42);

  softmax<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size1 * size2 * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
