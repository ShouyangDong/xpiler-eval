#include "hip/hip_runtime.h"
// Kernel: reduce along axis=1 for input [16, 32, 32] -> output [16, 32]
// Each thread handles one (n, w) position
__global__ void max_dev(const float* __restrict__ input, float* __restrict__ output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int output_size = 16 * 32;  // 512
    if (idx >= output_size) return;

    int n = idx / 32;  // batch index: 0~15
    int w = idx % 32;  // width index: 0~31

    float max_val = -FLT_MAX;
    for (int h = 0; h < 32; h++) {
        int in_idx = n * (32 * 32) + h * 32 + w;  // input[n][h][w]
        max_val = fmaxf(max_val, input[in_idx]);
    }
    output[idx] = max_val;
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C" void max_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 16 * 32 * 32;  // 16384
        const int output_size = 16 * 32;      // 512

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(256);
        dim3 numBlocks((output_size + 255) / 256);  // (512 + 255) / 256 = 3

        max_dev<<<numBlocks, blockSize>>>(d_input, d_output);

        // Copy result back to host
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_input);
        hipFree(d_output);
}