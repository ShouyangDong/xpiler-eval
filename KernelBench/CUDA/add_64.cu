
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(64)
    add(float *__restrict__ A, float *__restrict__ B,
        float *__restrict__ T_add) {
  T_add[((int)threadIdx.x)] = (A[((int)threadIdx.x)] + B[((int)threadIdx.x)]);
}

extern "C" void add_kernel(float *A, float *B, float *C, int size) {
  float *d_A;
  float *d_B;
  float *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_B, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(64);
  dim3 numBlocks((size + 64 - 1) / 64);

  add<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
