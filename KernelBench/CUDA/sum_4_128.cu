
#include <hip/hip_runtime.h>


__global__ void sum_kernel_dev(const float *__restrict__ input,
                               float *__restrict__ output) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= 4)
    return;

  float sum = 0.0f;
  for (int col = 0; col < 128; col++) {
    int idx = row * 128 + col;
    sum += input[idx];
  }
  output[row] = sum;
}

extern "C" void sum_kernel(const float *h_input, float *h_output) {
  float *d_input, *d_output;
  const int input_size = 4 * 128;
  const int output_size = 4;

  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, h_input, input_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(4);
  dim3 numBlocks(1);

  sum_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);

  hipMemcpy(h_output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}
