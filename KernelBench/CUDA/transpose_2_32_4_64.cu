
#include <hip/hip_runtime.h>

__global__ void transpose(const float* __restrict__ input,
                                 float* __restrict__ output) {
    const int N = 2;
    const int C = 32;
    const int H = 4;
    const int W = 64;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * H * W;
    if (idx < total) {
        // inline 方式计算 in/out 索引
        int out_idx =
            (idx / (C * H * W)) * (H * C * W) +     // n
            ((idx / W) % H) * (C * W) +             // h
            ((idx / (H * W)) % C) * W +             // c
            (idx % W);                              // w

        int in_idx =
            (idx / (C * H * W)) * (C * H * W) +     // n
            ((idx / (H * W)) % C) * (H * W) +       // c
            ((idx / W) % H) * W +                   // h
            (idx % W);                              // w

        output[out_idx] = input[in_idx];
    }
}

extern "C" void transpose_kernel(float* input, float* output,
                                 int N, int C, int H, int W) {


    int total = N * C * H * W;
    float *d_input, *d_output;
    hipMalloc(&d_input, total * sizeof(float));
    hipMalloc(&d_output, total * sizeof(float));

    hipMemcpy(d_input, input, total * sizeof(float), hipMemcpyHostToDevice);
    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    transpose<<<blocks, threads>>>(d_input, d_output);

    hipMemcpy(output, d_output, total * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
