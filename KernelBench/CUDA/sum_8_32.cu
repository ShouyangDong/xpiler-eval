// Generated: sum along last dimension for input [8x16x32x32] -> [8x16x32]
// Total input: 131072, Reduce size: 32, Output count: 4096

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void __launch_bounds__(256)
sum(const float *__restrict__ input, float *__restrict__ output) {
    int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= 4096) return;

    float sum = 0.0f;
    for (int i = 0; i < 32; i++) {
        int in_idx = out_idx * 32 + i;
        sum += input[in_idx];
    }
    output[out_idx] = sum;
}

extern "C" void  sum_kernel(const float *h_input, float *h_output) {
    float *d_input, *d_output;
    const int input_size = 131072;
    const int output_size = 4096;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 numBlocks((output_size + 255) / 256);

    sum<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
