#include "hip/hip_runtime.h"
// Kernel: reduce along axis=0 for input [64] -> output scalar
// Only one thread needed, but we use one block for simplicity
__global__ void min_kernel_dev(const float* __restrict__ input, float* __restrict__ output) {
    float min_val = FLT_MAX;  // Initialize to +infinity

    for (int i = 0; i < 64; i++) {
        min_val = fminf(min_val, input[i]);
    }

    *output = min_val;
}

// Host wrapper - DO NOT CHANGE FUNCTION NAME
extern "C"
    void min_kernel(const float* h_input, float* h_output) {
        float *d_input, *d_output;
        const int input_size = 64;
        const int output_size = 1;  // scalar

        // Allocate device memory
        hipMalloc(&d_input, input_size * sizeof(float));
        hipMalloc(&d_output, output_size * sizeof(float));

        // Copy input from host to device
        hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel
        // Only need 1 thread, but use a small block
        dim3 blockSize(1);
        dim3 numBlocks(1);

        min_kernel_dev<<<numBlocks, blockSize>>>(d_input, d_output);


        // Copy result back to host (h_output points to a single float)
        hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_input);
        hipFree(d_output);

}
